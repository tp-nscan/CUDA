#include "../../common/common.h"
#include "../../common/book.h"
#include "../../common/DimStuff.h"
#include "../../common/Rando.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>


void RandBench();


int main()
{
	RandBench();
}


void RandBench()
{
	RandData data;
	data.blocks = dim3(128);
	data.threads = dim3(256);
	data.seed = 123;

	hiprandState *dev_rngs = data.dev_rngs;
	hipEvent_t     start, stop;
	float totalTime = 0;
	float frames = 0;
	int arrayLength = 1000000;
	int reps = 100;

	hipError_t cudaResult = hipSuccess;

	float *dev_rands;
	float *host_rands;
	float   elapsedTime;

	cudaResult = Gpu_InitRNG(&dev_rngs, &data);


	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	HANDLE_ERROR(hipEventRecord(start, 0));

	for (int i = 0; i < reps; i++) {

		Gpu_UniformRandFloats(&dev_rands, &data, arrayLength);
	}

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	totalTime = elapsedTime;
	printf("Uniform:  %3.1f ms\n", totalTime); // / data->frames);

	 
	HANDLE_ERROR(hipEventRecord(start, 0));

	for (int i = 0; i < reps; i++) {

		Gpu_NormalRandFloats(&dev_rands, &data, arrayLength);
	}

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	totalTime = elapsedTime;
	printf("Normal:  %3.1f ms\n", totalTime); // / data->frames);




	//Gpu_GetFloats(&host_rands, dev_rands, data.arrayLength);

	//PrintFloatArray(host_rands, 10, data.arrayLength);

}
