#include "hip/hip_runtime.h"

#include "../../common/common.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>

hipError_t addWithCuda(int *c, int *a, int *b, dim3 size);

hipError_t addLongsWithCuda(long *c, long *a, long *b, unsigned long size);

void DoIntAdd();
void DoLongAdd();

void CompCpuGpu(dim3 *arraySize);
void DynLongAdd();

void DoCudaStuff();

__global__ void addKernel_T(int *c, int *a, int *b)
{
	int x = threadIdx.x;
	int y = threadIdx.y;
	int i = x + y; // * blockDim.x;
	c[i] = a[i] + b[i];
}

__global__ void addKernel_B(int *c, int *a, int *b)
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int i = x + y * gridDim.x;
	c[i] = a[i] + b[i];
}

__global__ void addLongKernel(long *c, long *a, long *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

void IntVector(int *vec, int ct, int mult, int offset)
{
	for (int i = 0; i < ct; i++)
	{
		vec[i] = offset + i * mult;
	}
}

void LongVector(long *vec, long ct, long mult, long offset)
{
	for (int i = 0; i < ct; i++)
	{
		vec[i] = offset + i * mult;
	}
}


dim3 *dim3Ctr(int x, int y = 1, int z = 1)
{
	dim3 *a;
	a = (dim3 *)malloc(sizeof(dim3));
	a->x = x;
	a->y = y;
	a->z = z;
	return a;
}


void printDim3(dim3 *yow)
{
	printf("yow: {%d, %d, %d}", yow->x, yow->y, yow->z);
}

int main()
{
	DoCudaStuff();
    return 0;
}


void DoCudaStuff()
{
	//int ck = 6400;
	int ck = 400;
	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 4; j++)
		{

			dim3 *td = dim3Ctr(ck + ck * i, ck + ck * j);
			CompCpuGpu(td);
		}
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	CHECK(hipDeviceReset());
}

int *Gpu_R2by1(dim3 *arraySize, int *av, int *bv)
{
	int arrayLength = arraySize->x * arraySize->y;
	int *c =  (int *)malloc(sizeof(int) * arrayLength);

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, av, bv, *arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		exit(1);
	}

	return c;
}

int *Cpu_R2by1(dim3 *arraySize, int *av, int *bv)
{
	int arrayLength = arraySize->x * arraySize->y;
	int *c = (int *)malloc(sizeof(int) * arrayLength);

	for (int i = 0; i < arrayLength; i++)
	{
		c[i] = av[i] + bv[i];
	}
	return c;
}

bool CompArrays(int *a, int *b, int length)
{
	for (int i = 0; i < length; i++)
	{
		if (a[i] != b[i]) return false;
	}
	return true;
}

void CompCpuGpu(dim3 *arraySize)
{
	int arrayLength = arraySize->x * arraySize->y;

	int *av = (int *)malloc(sizeof(int) * arrayLength);
	int *bv = (int *)malloc(sizeof(int) * arrayLength);
	IntVector(av, arrayLength, 1, 1);
	IntVector(bv, arrayLength, 10, 0);

	int *cGPU = Gpu_R2by1(arraySize, av, bv);
	int *cCPU = Cpu_R2by1(arraySize, av, bv);

	printf("arrayLength: %d ", arrayLength);
	CompArrays(cCPU, cGPU, arrayLength) ? printf("pass\n") : printf("fail\n");
	
	free(cGPU);
	free(cCPU);
	free(av);
	free(bv);
}

void DynLongAdd()
{
	const long arraySize = 500;
	long c[arraySize] = { 0 };
	long av[arraySize], bv[arraySize];
	LongVector(av, arraySize, 1, 1);
	LongVector(bv, arraySize, 10, 0);

	// Add vectors in parallel.
	hipError_t cudaStatus = addLongsWithCuda(c, av, bv, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		exit(1);
	}

	printf("first five terms of c = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);
}


void DoIntAdd()
{
	const int arraySize = 50;
	int c[arraySize] = { 0 };
	int av[arraySize], bv[arraySize];
	IntVector(av, arraySize, 1, 1);
	IntVector(bv, arraySize, 10, 0);

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, av, bv, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		exit(1);
	}

	printf("first five terms of c = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);
}

void DoLongAdd()
{
	const long arraySize = 500;
	long c[arraySize] = { 0 };
	long av[arraySize], bv[arraySize];
	LongVector(av, arraySize, 1, 1);
	LongVector(bv, arraySize, 10, 0);

	// Add vectors in parallel.
	hipError_t cudaStatus = addLongsWithCuda(c, av, bv, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		exit(1);
	}

	printf("first five terms of c = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addLongsWithCuda(long *c, long *a, long *b, unsigned long size)
{
	 long *dev_a = 0;
	 long *dev_b = 0;
	 long *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(long), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(long), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addLongKernel <<<1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(long), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, int *a, int *b, dim3 arraySize)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

	int arrayLength = arraySize.x * arraySize.y;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, arrayLength * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, arrayLength * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, arrayLength * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, arrayLength * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, arrayLength * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	addKernel_T <<<1, arraySize>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, arrayLength * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
