#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

int *IntArray(int length, int first = 0, int step = 0)
{
	int *av = (int *)malloc(sizeof(int) * length);
	for (int i = 0; i < length; i++)
	{
		av[i] = first + step * i;
	}
	return av;
}

bool CompIntArrays(int *a, int *b, int length)
{
	for (int i = 0; i < length; i++)
	{
		if (a[i] != b[i]) return false;
	}
	return true;
}

bool CompFloatArrays(float *a, float *b, int length)
{
	for (int i = 0; i < length; i++)
	{
		if (a[i] != b[i]) return false;
	}
	return true;
}

void PrintFloatArray(float *aa, int width, int length)
{
	for (int i = 0; i < length; i++) {
		printf("%3.3f ", aa[i]);
		if ((i>0) && ((i + 1) % width == 0)) printf("\n");
	}
	printf("\n");
}

void PrintIntArray(int *aa, int width, int length)
{
	for (int i = 0; i < length; i++) {
		printf("%d ", aa[i]);
		if ((i>0) && ((i + 1) % width == 0)) printf("\n");
	}
	printf("\n");
}

float *RndFloat0to1(int arraySize)
{
	float *temp = (float*)malloc(arraySize * sizeof(float));
	for (int i = 0; i<arraySize; i++) {
		temp[i] = (float)rand() / (float)(RAND_MAX);
	}
	return temp;
}
