#include "hip/hip_runtime.h"
#include "common.h"
#include "DimStuff.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "Rando.h"

__global__ void initRNG(hiprandState *const rngStates, const unsigned int seed)
{
	// Determine thread ID
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	// Initialise the RNG
	hiprand_init(seed, tid, 0, &rngStates[tid]);
}

// Estimator kernel
//template <typename Real>
__global__ void gen_uniform(float *const results, hiprandState *const rngStates,
	const unsigned int numRands)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int step = gridDim.x * blockDim.x;
	for (unsigned int i = tid; i < numRands; i += step)
	{
		hiprandState localState = rngStates[tid];
		results[i] = hiprand_uniform(&localState);
	}
}

__global__ void gen_normal(float *const results, hiprandState *const rngStates,
	const unsigned int numRands)
{
	//// Determine thread ID
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int step = gridDim.x * blockDim.x;

	for (unsigned int i = tid; i < numRands; i += step)
	{
		hiprandState localState = rngStates[tid];
		results[i] = hiprand_uniform(&localState);
	}
}


__device__ inline void getPoint(float &x, float &y, hiprandState &state)
{
	x = hiprand_uniform(&state);
	y = hiprand_uniform(&state);
}

__device__ inline void getPoint(double &x, double &y, hiprandState &state)
{
	x = hiprand_uniform_double(&state);
	y = hiprand_uniform_double(&state);
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// creates and fills a device pointer with the curandStates
hipError_t Gpu_InitRNG(hiprandState **dev_rngs, RandData *randData)
{
	hipError_t cudaStatus = hipSuccess;
	CHECK_G(hipSetDevice(0));

	hiprandState *dev_rngens;
	CHECK_G(hipMalloc((void**)&dev_rngens, dim3Vol(&randData->blocks) * dim3Vol(&randData->threads) * sizeof(hiprandState)));
	randData->dev_rngs = dev_rngens;

	initRNG << <randData->blocks, randData->threads >> >(randData->dev_rngs, randData->seed);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	CHECK_G(hipDeviceSynchronize());

	return cudaStatus;
Error:
	hipFree(randData->dev_rngs);
	return cudaStatus;
}

// creates and fills a device pointer with the curandStates
hipError_t Gpu_UniformRandFloats(float **dev_rands, RandData *randData, int numRands)
{
	hipError_t cudaStatus = hipSuccess;
	CHECK_G(hipSetDevice(0));

	float *dev_r;
	cudaStatus = hipMalloc((void**)&dev_r, numRands * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	randData->dev_rngs;

	gen_uniform << <randData->blocks, randData->threads >> >(dev_r, randData->dev_rngs, numRands);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	CHECK_G(hipDeviceSynchronize());
	*dev_rands = dev_r;

Error:
	return cudaStatus;
}


// creates and fills a device pointer with the curandStates
hipError_t Gpu_NormalRandFloats(float **dev_rands, RandData *randData,	int numRands)
{
	hipError_t cudaStatus = hipSuccess;
	CHECK_G(hipSetDevice(0));

	float *dev_r;
	cudaStatus = hipMalloc((void**)&dev_r, numRands * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	randData->dev_rngs;

	gen_normal << <randData->blocks, randData->threads >> >(dev_r, randData->dev_rngs, numRands);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	CHECK_G(hipDeviceSynchronize());
	*dev_rands = dev_r;

Error:
	return cudaStatus;
}