#pragma once
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>


dim3 *dim3Ctr(int x, int y = 1, int z = 1)
{
	dim3 *a;
	a = (dim3 *)malloc(sizeof(dim3));
	a->x = x;
	a->y = y;
	a->z = z;
	return a;
}

dim3 *dim3Unit()
{
	dim3 *a;
	a = (dim3 *)malloc(sizeof(dim3));
	a->x = 1;
	a->y = 1;
	a->z = 1;
	return a;
}

int dim3Vol(dim3 *a)
{
	return a->x * a->y * a->z;
}

void printDim3(dim3 *yow)
{
	printf("yow: {%d, %d, %d}", yow->x, yow->y, yow->z);
}

