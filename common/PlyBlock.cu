#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"
#include ""
#include "common.h"
#include "PlyBlock.h"
#include "Rando.h"
#include "TexUtils.h"

void MakePlyBlock(PlyBlock **out, unsigned int width, unsigned int seed)
{
	PlyBlock *pbRet = (PlyBlock *)malloc(sizeof(PlyBlock));;
	*out = pbRet;

	RandData *randData = (RandData *)malloc(sizeof(RandData));;

	pbRet->width = width;
	pbRet->area = width * width;
	pbRet->randData = randData;

	unsigned int plyMemSize = pbRet->area * sizeof(float);
	hipError_t cudaResult = hipSuccess;

	CHECK(hipMalloc((void**)&pbRet->dev_inSrc, plyMemSize));
	CHECK(hipMalloc((void**)&pbRet->dev_outSrc, plyMemSize));
	//CHECK(hipMalloc((void**)&pbRet->dev_constSrc, plyMemSize));

	pbRet->texIn = TexObjFloat1D(pbRet->dev_inSrc, plyMemSize);
	pbRet->texOut = TexObjFloat1D(pbRet->dev_outSrc, plyMemSize);
	//pbRet->texConst = TexObjFloat1D(pbRet->dev_constSrc, plyMemSize);

	int chop = 64;
	randData->seed = seed;
	randData->blocks = dim3(width/chop, width/chop);
	randData->threads = dim3(chop/4, chop/4);
	CHECK(Gpu_InitRNG_2d(randData));
}