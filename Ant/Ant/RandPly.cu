#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "Utils.h"
#include "BlockUtils.h"
#include "Rando.h"
#include "Ply.h"
#include "RandPly.h"

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) {printf("Error at %s:%d\n",__FILE__,__LINE__); return EXIT_FAILURE;}} while(0)


////////////////////////////////////////////////////////////////////////////////
// randPly_local_update
// A 1d texture, working on a 2d torus, using 1d blocks and threads
////////////////////////////////////////////////////////////////////////////////
__global__ void randPly_local_update(float *resOut, float *plyIn, float *rands, int2 plySize,
	float speed, float noise) {

	int plyLength = plySize.x * plySize.y;
	int step = blockDim.x * gridDim.x;

	for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < plyLength; i += step)
	{
		int col = i % plySize.x;
		int row = (i - col) / plySize.x;
		int colm = (col - 1 + plySize.x) % plySize.x;
		int colp = (col + 1 + plySize.x) % plySize.x;
		int rowm = (row - 1 + plySize.y) % plySize.y;
		int rowp = (row + 1 + plySize.y) % plySize.y;

		int left = colm + row * plySize.x;
		int right = colp + row * plySize.x;
		int top = col + rowp * plySize.x;
		int bottom = col + rowm * plySize.x;
		int center = col + row * plySize.x;

		float t = plyIn[top];
		float l = plyIn[left];
		float c = plyIn[center];
		float r = plyIn[right];
		float b = plyIn[bottom];

		float res = c + speed * (t + b + r + l) + noise * rands[i];
		if (res > 1.0) {
			res = 1.0;
		}
		if (res < -1.0) {
			res = -1.0;
		}
		resOut[center] = res;
	}
}

////////////////////////////////////////////////////////////////////////////////
//! MakeRandPly
////////////////////////////////////////////////////////////////////////////////
void MakeRandPly(RandPly **randPly, float *data, int seed, unsigned int plyLength, unsigned int span)
{
	RandPly *rp = (RandPly *)malloc(sizeof(RandPly));
	*randPly = rp;
	MakePly(&(rp->ply), data, plyLength, span);
	float *dev_rands;
	int dataSize = plyLength * sizeof(float);
	checkCudaErrors(hipMalloc((void **)&dev_rands, dataSize));
	InitRandData(&(rp->randData), seed, plyLength, dev_rands);
}

////////////////////////////////////////////////////////////////////////////////
//! RunRandPlyLocalUpdate
////////////////////////////////////////////////////////////////////////////////
void RunRandPlyLocalUpdate(RandPly *randPly, int num_steps, float speed, float noise)
{
	int blocks = SuggestedBlocks((randPly->ply->area + THREADS_1D - 1) / THREADS_1D);
	int2 plySize;
	plySize.x = randPly->ply->span; plySize.y = randPly->ply->span;

	for (int step = 0; step < num_steps; step++)
	{
		UpdateRandData(randPly->randData);

		if (randPly->ply->inToOut)
		{
			randPly_local_update << <blocks, THREADS_1D >> >(
				randPly->ply->dev_outSrc,
				randPly->ply->dev_inSrc,
				randPly->randData->dev_rands,
				plySize,
				speed,
				noise);
		}
		else
		{
			randPly_local_update << <blocks, THREADS_1D >> >(
				randPly->ply->dev_inSrc,
				randPly->ply->dev_outSrc,
				randPly->randData->dev_rands,
				plySize,
				speed,
				noise);
		}

		randPly->ply->inToOut = !randPly->ply->inToOut;
	}
}

////////////////////////////////////////////////////////////////////////////////
//! GridEnergy
////////////////////////////////////////////////////////////////////////////////
float *GridEnergy(float *d_ply, int span, int dataLen)
{
	int blocks = SuggestedBlocks((dataLen + THREADS_1D - 1) / THREADS_1D);

	int plyMemSize = span * span * sizeof(float);
	int2 plySize;
	plySize.x = span; plySize.y = span;

	float *d_energies;
	checkCudaErrors(hipMalloc((void**)&d_energies, plyMemSize));

	grid_local_energy << <blocks, THREADS_1D >> >(
		d_energies,
		d_ply,
		plySize);

	float *h_energies = (float *)malloc(plyMemSize);
	checkCudaErrors(hipMemcpy(h_energies, d_energies, plyMemSize, hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(d_energies));
	return h_energies;
}

////////////////////////////////////////////////////////////////////////////////
//! GridEnergyC
////////////////////////////////////////////////////////////////////////////////
float *GridEnergyC(float *d_ply, int span, int dataLen)
{
	int blocks = SuggestedBlocks((dataLen + THREADS_1D - 1) / THREADS_1D);

	int blockEnergySize = blocks * blocks * sizeof(float);
	int2 plySize;
	plySize.x = span; plySize.y = span;

	float *d_energies;
	checkCudaErrors(hipMalloc((void**)&d_energies, blockEnergySize));

	grid_local_energyC << <blocks, THREADS_1D >> >(
		d_energies,
		d_ply,
		plySize);

	float *h_energies = (float *)malloc(blockEnergySize);
	checkCudaErrors(hipMemcpy(h_energies, d_energies, blockEnergySize, hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(d_energies));
	return h_energies;
}

////////////////////////////////////////////////////////////////////////////////
//! PlyEnergy
////////////////////////////////////////////////////////////////////////////////
float *PlyEnergy(Ply *ply)
{
	if (ply->inToOut)
	{
		return GridEnergy(ply->dev_inSrc, ply->span, ply->area);
	}
	else
	{
		return GridEnergy(ply->dev_outSrc, ply->span, ply->area);
	}
}


////////////////////////////////////////////////////////////////////////////////
//! PlyEnergyC
////////////////////////////////////////////////////////////////////////////////
float *PlyEnergyC(Ply *ply)
{
	if (ply->inToOut)
	{
		return GridEnergyC(ply->dev_inSrc, ply->span, ply->area);
	}
	else
	{
		return GridEnergyC(ply->dev_outSrc, ply->span, ply->area);
	}
}

////////////////////////////////////////////////////////////////////////////////
//! DeleteRandPly
////////////////////////////////////////////////////////////////////////////////
void DeleteRandPly(RandPly *randPly)
{
	FreePly(randPly->ply);
	DeleteRandData(randPly->randData);
}


////////////////////////////////////////////////////////////////////////////////
//! RunEnergyTest
////////////////////////////////////////////////////////////////////////////////
void RunEnergyTest(int argc, char **argv)
{
	//dataLen=400 span=20 reps=50 seed=1243 speed=0.05 noise=0.01 batch=10
	int dataLen = IntNamed(argc, argv, "dataLen", 81);
	int span = IntNamed(argc, argv, "span", 9);

	float *d_pattern;
	float *h_pattern = CheckerArray(span);
	PrintFloatArray(h_pattern, span, span*span);

	int plyMemSize = span * span * sizeof(float);

	checkCudaErrors(hipMalloc((void**)&d_pattern, plyMemSize));
	checkCudaErrors(hipMemcpy(d_pattern, h_pattern, plyMemSize, hipMemcpyHostToDevice));

	float *h_energies = GridEnergy(d_pattern, span, dataLen);

	printf("Energies:\n");
	PrintFloatArray(h_energies, span, span*span);
}


////////////////////////////////////////////////////////////////////////////////
//! RunEnergyTestC
////////////////////////////////////////////////////////////////////////////////
void RunEnergyTestC(int argc, char **argv)
{
	//dataLen=400 span=20 reps=50 seed=1243 speed=0.05 noise=0.01 batch=10
	int dataLen = IntNamed(argc, argv, "dataLen", 81);
	int span = IntNamed(argc, argv, "span", 9);


	int blocks = SuggestedBlocks((dataLen + THREADS_1D - 1) / THREADS_1D);

	float *d_pattern;
	float *h_pattern = CheckerArray(span);
	PrintFloatArray(h_pattern, span, span*span);

	int plyMemSize = span * span * sizeof(float);

	checkCudaErrors(hipMalloc((void**)&d_pattern, plyMemSize));
	checkCudaErrors(hipMemcpy(d_pattern, h_pattern, plyMemSize, hipMemcpyHostToDevice));

	float *h_energies = GridEnergyC(d_pattern, span, dataLen);

	//float *h_energies = (float *)malloc(plyMemSize);
	//checkCudaErrors(hipMemcpy(h_energies, d_pattern, plyMemSize, hipMemcpyDeviceToHost));

	printf("Energies:\n");
	PrintFloatArray(h_energies, 1, blocks);
}

float *RunRandPly(int span, float speed, float noise, int seed, float *h_data, int reps)
{
	RandPly *randPly;
	MakeRandPly(&(randPly), h_data, seed, span*span, span);
	RunRandPlyLocalUpdate(randPly, reps, speed, noise);
	float *h_results = (float *)malloc(span * span * sizeof(float));
	GetPlyData(randPly->ply, h_results);
	DeleteRandPly(randPly);

	return h_results;
}

////////////////////////////////////////////////////////////////////////////////
//! RunRandPlyBench
////////////////////////////////////////////////////////////////////////////////
void RunRandPlyBench(int argc, char **argv)
{
	//dataLen=1048576 span=1024 reps=100 seed=6423 speed=0.1 noise=0.01 batch=1
	//dataLen=400 span=20 reps=50 seed=1243 speed=0.05 noise=0.01 batch=10
	int dataLen = IntNamed(argc, argv, "dataLen", 36);
	int span = IntNamed(argc, argv, "span", 6);
	int seed = IntNamed(argc, argv, "seed", 12);
	int reps = IntNamed(argc, argv, "reps", 16);
	float speed = FloatNamed(argc, argv, "speed", 0.05);
	float noise = FloatNamed(argc, argv, "noise", 0.01);
	int batch = IntNamed(argc, argv, "batch", 10);
	printf("dataLen: %d  batch: %d  speed: %3.4f  noise: %3.4f \n", dataLen, batch, speed, noise);

	float *h_samples;

	int dataSize = dataLen * sizeof(float);
	h_samples = CheckerArray(span);

	RandPly *randPly;
	MakeRandPly(&(randPly), h_samples, seed, dataLen, span);

	hipEvent_t start, stop;
	float   elapsedTime;

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	for (int i = 0; i < reps; i++) {
		RunRandPlyLocalUpdate(randPly, batch, speed, noise);		
		float *out = PlyEnergy(randPly->ply);
		printf("Energy:  %3.4f\n", FloatArraySum(out, dataLen));
	}

	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

	printf("Time:  %3.1f ms\n", elapsedTime);
	GetPlyData(randPly->ply, h_samples);
	//PrintFloatArray(h_samples, span, dataLen);

	DeleteRandPly(randPly);
}


