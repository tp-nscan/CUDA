#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "Utils.h"
#include "BlockUtils.h"
#include "Rando.h"
#include <ctime>

////////////////////////////////////////////////////////////////////////////////
//! RandData Initialization
////////////////////////////////////////////////////////////////////////////////

__global__ void initRNG(hiprandState *const rngStates, const unsigned int seed,
	const unsigned int length)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int step = gridDim.x * blockDim.x;

	for (unsigned int i = tid; i < length; i += step)
	{
		hiprand_init(seed, tid, 0, &rngStates[tid]);
		//hiprand_init((unsigned long long)clock() + i, 0, 0, &rngStates[tid]);
	}
}


void InitRandData(RandData **out, int seed, int dataLen, float *dev_rands)
{
	hiprandStatus_t status;

	RandData *randData = (RandData *)malloc(sizeof(RandData));
	*out = randData;
	randData->index = 0;
	randData->seed = seed;
	randData->length = dataLen;
	randData->dev_rands = dev_rands;
	int dataSize = dataLen * sizeof(float);

	status = hiprandCreateGenerator(&(randData->rand_gen), HIPRAND_RNG_PSEUDO_XORWOW);
	hiprandSetPseudoRandomGeneratorSeed(randData->rand_gen, seed);

	getLastCudaError("InitRandData execution failed");
}


void UpdateRandData(RandData *randData)
{
	//hiprandGenerateUniform(randData->rand_gen, randData->dev_rands, randData->length);
	hiprandGenerateNormal(randData->rand_gen, randData->dev_rands, randData->length, 0.0, 1.0);
	randData->index++;
	getLastCudaError("MakeRandData execution failed");
}


void DeleteRandData(RandData *randData)
{
	hipFree(randData->dev_rands);
	hiprandDestroyGenerator(randData->rand_gen);
	free(randData);
}


////////////////////////////////////////////////////////////////////////////////
// Rand Generation
////////////////////////////////////////////////////////////////////////////////


__global__ void rndGenNormal(float *const results,  hiprandState *const rngStates,
                                const unsigned int numRands)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int step = gridDim.x * blockDim.x;

	for (unsigned int i = tid; i < numRands; i += step)
	{
		hiprandState localState = rngStates[tid];
		results[i] = hiprand_normal(&localState);
	}
}

__global__ void rndGenUniform(float *const results, hiprandState *const rngStates, 
	                             const unsigned int numRands)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int step = gridDim.x * blockDim.x;

	for (unsigned int i = tid; i < numRands; i += step)
	{
		hiprandState localState = rngStates[tid];
		results[i] = hiprand_uniform(&localState);
	}
}


__device__ inline void getPoint(float &x, float &y, hiprandState &state)
{
	x = hiprand_uniform(&state);
	y = hiprand_uniform(&state);
}

__device__ inline void getPoint(double &x, double &y, hiprandState &state)
{
	x = hiprand_uniform_double(&state);
	y = hiprand_uniform_double(&state);
}


////////////////////////////////////////////////////////////////////////////////
//! Make and test RandData
////////////////////////////////////////////////////////////////////////////////
void RandTest(int argc, char **argv)
{
	//dataLen=10 reps=10 seed=1243
	int seed = IntNamed(argc, argv, "seed", 1234);
	int dataLen = IntNamed(argc, argv, "dataLen", 512);
	int reps = IntNamed(argc, argv, "reps", 512);

	float *dev_data;
	int dataSize = dataLen * sizeof(float);
	checkCudaErrors(hipMalloc((void **)&dev_data, dataSize));

	RandData *randData;
	InitRandData(&randData, seed, dataLen, dev_data);

	float *h_samples = (float *)malloc(dataSize);

	hipEvent_t start, stop;

	float *dev_rands;
	float *host_rands;
	float   elapsedTime;

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	checkCudaErrors(hipEventRecord(start, 0));

	for (int i = 0; i < reps; i++) {
		UpdateRandData(randData);
		checkCudaErrors(hipMemcpy(h_samples, randData->dev_rands, dataSize, hipMemcpyDeviceToHost));
		PrintFloatArray(h_samples, 1, dataLen);
	}

	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Uniform:  %3.1f ms\n", elapsedTime);

	DeleteRandData(randData);
}
